/*-
 * Copyright 2015 Grammarly, Inc.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <hip/hip_runtime.h>


__global__ void testDependencies(int node_id, int* blocking_nodes, int blocking_nodes_num, int *execution_checklist, int* test_results) {
	test_results[node_id] = 1;
	for (int i = 0; i < blocking_nodes_num; i++) {
		int bloking_node_id = blocking_nodes[i];
		if (!execution_checklist[bloking_node_id]) {
			test_results[node_id] = 0;
			break;
		}
	}

	clock_t start_clock = clock64();
    clock_t clock_offset = 0;
    while (clock_offset < 4000000000L) {
        clock_offset = clock64() - start_clock;
    }

    execution_checklist[node_id] = 1;
}


extern "C" {
    hipError_t _testDependencies(hipStream_t stream, int node_id, int* blocking_nodes, int blocking_nodes_num, int *execution_checklist, int* test_results) {
		testDependencies<<<1, 1, 0, stream>>>(node_id, blocking_nodes, blocking_nodes_num, execution_checklist, test_results);
		return hipGetLastError();
    }
}