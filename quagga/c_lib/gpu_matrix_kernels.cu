#include <algorithm>
#include <hip/hip_runtime.h>


#define MAX_NUM_THREADS_PER_BLOCK 512
#define MAX_NUM_BLOCKS_PER_KERNEL 64


__global__  void sliceColumns(int nrows,
							  int ncols,
							  const int* __restrict__ embedding_column_indxs,
							  const float* __restrict__ embedding_matrix,
							  float* __restrict__ dense_matrix) {
	const int nthreads = blockDim.x * gridDim.x;
	const int start_i = blockIdx.x * blockDim.x + threadIdx.x;
	const int nelems = nrows * ncols;

	int dense_column_idx;
	int row_idx;
	int embedding_offset;
	for (int i = start_i; i < nelems; i += nthreads) {
		dense_column_idx = i / nrows;
		row_idx = i % nrows;
		embedding_offset = embedding_column_indxs[dense_column_idx] * nrows + row_idx;
		dense_matrix[i] = embedding_matrix[embedding_offset];
	}
}


__global__ void testDependencies(int node_id, int* blocking_nodes, int blocking_nodes_num, int *execution_checklist, int* test_results) {
	test_results[node_id] = 1;
	for (int i = 0; i < blocking_nodes_num; i++) {
		int bloking_node_id = blocking_nodes[i];
		if (!execution_checklist[bloking_node_id]) {
			test_results[node_id] = 0;
			break;
		}
	}

	clock_t start_clock = clock64();
    clock_t clock_offset = 0;
    while (clock_offset < 4000000000L) {
        clock_offset = clock64() - start_clock;
    }

    execution_checklist[node_id] = 1;
}


__global__ void fill(int nelems, float val, float* __restrict__ A) {
	const int nthreads = blockDim.x * gridDim.x;
	const int start_i = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = start_i; i < nelems; i += nthreads) {
		A[i] = val;
	}
}

__global__ void hprodSum(int nelems,
						 int nrows,
						 const float* __restrict__ A,
						 const float* __restrict__ B,
						 float* __restrict__ C) {
	const int nthreads = blockDim.x * gridDim.x;
	const int start_i = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = start_i; i < nelems; i += nthreads) {
		atomicAdd(C + i % nrows, A[i] * B[i]);
	}
}


__global__ void sumHprod(int nelems,
						 const float* __restrict__ A,
						 const float* __restrict__ B,
						 const float* __restrict__ C,
						 const float* __restrict__ D,
						 float* __restrict__ E) {
	const int nthreads = blockDim.x * gridDim.x;
	const int start_i = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = start_i; i < nelems; i += nthreads) {
		E[i] = A[i] * B[i] + C[i] * D[i];
	}
}


__global__ void sumHprod(int nelems,
						 const float* __restrict__ A,
						 const float* __restrict__ B,
						 const float* __restrict__ C,
						 const float* __restrict__ D,
						 const float* __restrict__ E,
						 float* __restrict__ F) {
	const int nthreads = blockDim.x * gridDim.x;
	const int start_i = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = start_i; i < nelems; i += nthreads) {
		F[i] = A[i] * B[i] * C[i] + D[i] * E[i];
	}
}


__global__ void sumHprod(int nelems,
						 const float* __restrict__ A,
						 const float* __restrict__ B,
						 const float* __restrict__ C,
						 const float* __restrict__ D,
						 const float* __restrict__ E,
						 const float* __restrict__ F,
						 const float* __restrict__ G,
						 const float* __restrict__ H,
						 const float* __restrict__ I,
						 const float* __restrict__ J,
						 const float* __restrict__ K,
						 float* __restrict__ L) {
	const int nthreads = blockDim.x * gridDim.x;
	const int start_i = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = start_i; i < nelems; i += nthreads) {
		L[i] = A[i] * B[i] * C[i] + D[i] * E[i] + F[i] * G[i] + H[i] * I[i] + J[i] * K[i];
	}
}


__global__  void hadamardProduct(int nelems,
							     const float* __restrict__ a,
							     const float* __restrict__ b,
							     float* __restrict__ c) {
	const int nthreads = blockDim.x * gridDim.x;
	const int start_i = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = start_i; i < nelems; i += nthreads) {
		c[i] = a[i] * b[i];
	}
}


__global__  void hadamardProduct(int nelems,
							     const float* __restrict__ a,
							     const float* __restrict__ b,
							     const float* __restrict__ c,
							     float* __restrict__ d) {
	const int nthreads = blockDim.x * gridDim.x;
	const int start_i = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = start_i; i < nelems; i += nthreads) {
		d[i] = a[i] * b[i] * c[i];
	}
}


__global__  void addHadamardProduct(int nelems,
							        const float* __restrict__ a,
							        const float* __restrict__ b,
							        float alpha,
							        float* __restrict__ c) {
	const int nthreads = blockDim.x * gridDim.x;
	const int start_i = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = start_i; i < nelems; i += nthreads) {
		c[i] = a[i] * b[i] + alpha * c[i];
	}
}


__global__  void slicedInplaceAdd(int nrows,
							      int ncols,
							      float alpha,
							      const float* __restrict__ dense_matrix,
							      const int* __restrict__ embedding_column_indxs,
							      float* __restrict__ embedding_matrix) {
	const int nthreads = blockDim.x * gridDim.x;
	const int start_i = blockIdx.x * blockDim.x + threadIdx.x;
	const int nelems = nrows * ncols;

	int dense_column_idx;
	int row_idx;
	int embedding_offset;
	for (int i = start_i; i < nelems; i += nthreads) {
		dense_column_idx = i / nrows;
		row_idx = i % nrows;
		embedding_offset = embedding_column_indxs[dense_column_idx] * nrows + row_idx;
		atomicAdd(embedding_matrix + embedding_offset, alpha * dense_matrix[i]);
	}
}


__global__ void sum(int nelems,
					const float* __restrict__ a,
					const float* __restrict__ b,
					const float* __restrict__ c,
					const float* __restrict__ d,
					float* __restrict__ e) {
	const int nthreads = blockDim.x * gridDim.x;
	const int start_i = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = start_i; i < nelems; i += nthreads) {
		e[i] = a[i] + b[i] + c[i] + d[i];
	}
}


__global__ void sigmoid(int nelems,
						const float* __restrict__ data,
						float* __restrict__ sigmoid_data) {
	const int nthreads = blockDim.x * gridDim.x;
	const int start_i = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = start_i; i < nelems; i += nthreads) {
		sigmoid_data[i] = 1.0f / (1.0f + expf(-data[i]));
	}
}


__global__ void sigmoid(int nelems,
						const float* __restrict__ data,
						float* __restrict__ sigmoid_data,
						float* __restrict__ derivative) {
	const int nthreads = blockDim.x * gridDim.x;
	const int start_i = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = start_i; i < nelems; i += nthreads) {
		sigmoid_data[i] = 1.0f / (1.0f + expf(-data[i]));
		derivative[i] = sigmoid_data[i] * (1.0f - sigmoid_data[i]);
	}
}


__global__ void tanh(int nelems,
					 const float* __restrict__ data,
					 float* __restrict__ tanh_data) {
	const int nthreads = blockDim.x * gridDim.x;
	const int start_i = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = start_i; i < nelems; i += nthreads) {
		tanh_data[i] = tanhf(data[i]);
	}
}


__global__ void tanh(int nelems,
					 const float* __restrict__ data,
					 float* __restrict__ tanh_data,
					 float* __restrict__ derivative) {
	const int nthreads = blockDim.x * gridDim.x;
	const int start_i = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = start_i; i < nelems; i += nthreads) {
		tanh_data[i] = tanhf(data[i]);
		derivative[i] = 1.0f - tanh_data[i] * tanh_data[i];
	}
}


__global__ void scale(int nelems,
					  const float* __restrict__ data,
					  float alpha,
					  float* __restrict__ out_data) {
	const int nthreads = blockDim.x * gridDim.x;
	const int start_i = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = start_i; i < nelems; i += nthreads) {
		out_data[i] = alpha * data[i];
	}
}


extern "C" {
	hipError_t _sliceColumns(hipStream_t stream,
							  int nrows,
							  int ncols,
							  const int* __restrict__ embedding_column_indxs,
							  const float* __restrict__ embedding_matrix,
							  float* __restrict__ dense_matrix) {
		int num_blocks = std::min(MAX_NUM_BLOCKS_PER_KERNEL, (nrows - 1) / MAX_NUM_THREADS_PER_BLOCK + 1);
		sliceColumns<<<num_blocks, MAX_NUM_THREADS_PER_BLOCK, 0, stream>>>(nrows, ncols, embedding_column_indxs, embedding_matrix, dense_matrix);
		return hipGetLastError();
	}


	hipError_t _testDependencies(hipStream_t stream, int node_id, int* blocking_nodes, int blocking_nodes_num, int *execution_checklist, int* test_results) {
		testDependencies<<<1, 1, 0, stream>>>(node_id, blocking_nodes, blocking_nodes_num, execution_checklist, test_results);
		return hipGetLastError();
	}


	hipError_t _hprodSum(hipStream_t stream,
                          int nrows,
                          int ncols,
						  const float* __restrict__ a,
						  const float* __restrict__ b,
						  float* __restrict__ c) {
		int num_blocks = std::min(MAX_NUM_BLOCKS_PER_KERNEL, (nrows - 1) / MAX_NUM_THREADS_PER_BLOCK + 1);
		fill<<<num_blocks, MAX_NUM_THREADS_PER_BLOCK, 0, stream>>>(nrows, 0.0, c);
		int nelems = nrows * ncols;
		num_blocks = std::min(MAX_NUM_BLOCKS_PER_KERNEL, (nelems - 1) / MAX_NUM_THREADS_PER_BLOCK + 1);
        hprodSum<<<num_blocks, MAX_NUM_THREADS_PER_BLOCK, 0, stream>>>(nelems, nrows, a, b, c);
        return hipGetLastError();
	}


    hipError_t _sumHprod4(hipStream_t stream,
                           int nelems,
						   const float* __restrict__ a,
						   const float* __restrict__ b,
						   const float* __restrict__ c,
						   const float* __restrict__ d,
						   float* __restrict__ e) {
		int num_blocks = std::min(MAX_NUM_BLOCKS_PER_KERNEL, (nelems - 1) / MAX_NUM_THREADS_PER_BLOCK + 1);
        sumHprod<<<num_blocks, MAX_NUM_THREADS_PER_BLOCK, 0, stream>>>(nelems, a, b, c, d, e);
        return hipGetLastError();
    }


    hipError_t _sumHprod5(hipStream_t stream,
                           int nelems,
						   const float* __restrict__ a,
						   const float* __restrict__ b,
						   const float* __restrict__ c,
						   const float* __restrict__ d,
						   const float* __restrict__ e,
						   float* __restrict__ f) {
        int num_blocks = std::min(MAX_NUM_BLOCKS_PER_KERNEL, (nelems - 1) / MAX_NUM_THREADS_PER_BLOCK + 1);
        sumHprod<<<num_blocks, MAX_NUM_THREADS_PER_BLOCK, 0, stream>>>(nelems, a, b, c, d, e, f);
        return hipGetLastError();
    }


    hipError_t _sumHprod11(hipStream_t stream,
                            int nelems,
						    const float* __restrict__ a,
						    const float* __restrict__ b,
						    const float* __restrict__ c,
						    const float* __restrict__ d,
						    const float* __restrict__ e,
						    const float* __restrict__ f,
						    const float* __restrict__ g,
						    const float* __restrict__ h,
						    const float* __restrict__ i,
						    const float* __restrict__ j,
						    const float* __restrict__ k,
						    float* __restrict__ l) {
        int num_blocks = std::min(MAX_NUM_BLOCKS_PER_KERNEL, (nelems - 1) / MAX_NUM_THREADS_PER_BLOCK + 1);
        sumHprod<<<num_blocks, MAX_NUM_THREADS_PER_BLOCK, 0, stream>>>(nelems, a, b, c, d, e, f, g, h, i, j, k, l);
        return hipGetLastError();
    }


    hipError_t _hadamardProduct2(hipStream_t stream,
                                  int nelems,
						     	  const float* __restrict__ a,
							      const float* __restrict__ b,
							      float* __restrict__ c) {
        int num_blocks = std::min(MAX_NUM_BLOCKS_PER_KERNEL, (nelems - 1) / MAX_NUM_THREADS_PER_BLOCK + 1);
        hadamardProduct<<<num_blocks, MAX_NUM_THREADS_PER_BLOCK, 0, stream>>>(nelems, a, b, c);
        return hipGetLastError();
    }


    hipError_t _hadamardProduct3(hipStream_t stream,
                                  int nelems,
						     	  const float* __restrict__ a,
							      const float* __restrict__ b,
							      const float* __restrict__ c,
							      float* __restrict__ d) {
        int num_blocks = std::min(MAX_NUM_BLOCKS_PER_KERNEL, (nelems - 1) / MAX_NUM_THREADS_PER_BLOCK + 1);
        hadamardProduct<<<num_blocks, MAX_NUM_THREADS_PER_BLOCK, 0, stream>>>(nelems, a, b, c, d);
        return hipGetLastError();
    }


    hipError_t _addHadamardProduct(hipStream_t stream,
                                    int nelems,
				 			        const float* __restrict__ a,
							        const float* __restrict__ b,
							        float alpha,
							        float* __restrict__ c) {
        int num_blocks = std::min(MAX_NUM_BLOCKS_PER_KERNEL, (nelems - 1) / MAX_NUM_THREADS_PER_BLOCK + 1);
        addHadamardProduct<<<num_blocks, MAX_NUM_THREADS_PER_BLOCK, 0, stream>>>(nelems, a, b, alpha, c);
        return hipGetLastError();
    }


    hipError_t _slicedInplaceAdd(hipStream_t stream,
                                  int nrows,
							      int ncols,
							      float alpha,
							      const float* __restrict__ dense_matrix,
							      const int* __restrict__ embedding_column_indxs,
							      float* __restrict__ embedding_matrix) {
        int num_blocks = std::min(MAX_NUM_BLOCKS_PER_KERNEL, (nrows * ncols - 1) / MAX_NUM_THREADS_PER_BLOCK + 1);
        slicedInplaceAdd<<<num_blocks, MAX_NUM_THREADS_PER_BLOCK, 0, stream>>>(nrows, ncols, alpha, dense_matrix, embedding_column_indxs, embedding_matrix);
        return hipGetLastError();
	}

    hipError_t _sum(hipStream_t stream,
                     int nelems,
					 const float* __restrict__ a,
					 const float* __restrict__ b,
					 const float* __restrict__ c,
					 const float* __restrict__ d,
					 float* __restrict__ e) {
		int num_blocks = std::min(MAX_NUM_BLOCKS_PER_KERNEL, (nelems - 1) / MAX_NUM_THREADS_PER_BLOCK + 1);
        sum<<<num_blocks, MAX_NUM_THREADS_PER_BLOCK, 0, stream>>>(nelems, a, b, c, d, e);
        return hipGetLastError();
	}


    hipError_t _sigmoid(hipStream_t stream,
                         int nelems,
			             const float* __restrict__ data,
			             float* __restrict__ sigmoid_data) {
	    int num_blocks = std::min(MAX_NUM_BLOCKS_PER_KERNEL, (nelems - 1) / MAX_NUM_THREADS_PER_BLOCK + 1);
        sigmoid<<<num_blocks, MAX_NUM_THREADS_PER_BLOCK, 0, stream>>>(nelems, data, sigmoid_data);
        return hipGetLastError();
	}


	hipError_t _sigmoid_der(hipStream_t stream,
                             int nelems,
			                 const float* __restrict__ data,
			                 float* __restrict__ sigmoid_data,
			                 float* __restrict__ derivative) {
	    int num_blocks = std::min(MAX_NUM_BLOCKS_PER_KERNEL, (nelems - 1) / MAX_NUM_THREADS_PER_BLOCK + 1);
        sigmoid<<<num_blocks, MAX_NUM_THREADS_PER_BLOCK, 0, stream>>>(nelems, data, sigmoid_data, derivative);
        return hipGetLastError();
	}


    hipError_t _tanh(hipStream_t stream,
                      int nelems,
			          const float* __restrict__ data,
			          float* __restrict__ tanh_data) {
	    int num_blocks = std::min(MAX_NUM_BLOCKS_PER_KERNEL, (nelems - 1) / MAX_NUM_THREADS_PER_BLOCK + 1);
        tanh<<<num_blocks, MAX_NUM_THREADS_PER_BLOCK, 0, stream>>>(nelems, data, tanh_data);
        return hipGetLastError();
	}


	hipError_t _tanh_der(hipStream_t stream,
                          int nelems,
			              const float* __restrict__ data,
			              float* __restrict__ tanh_data,
			              float* __restrict__ derivative) {
	    int num_blocks = std::min(MAX_NUM_BLOCKS_PER_KERNEL, (nelems - 1) / MAX_NUM_THREADS_PER_BLOCK + 1);
        tanh<<<num_blocks, MAX_NUM_THREADS_PER_BLOCK, 0, stream>>>(nelems, data, tanh_data, derivative);
        return hipGetLastError();
	}


	hipError_t _scale(hipStream_t stream,
	                   int nelems,
	                   float alpha,
                       const float* __restrict__ data,
                       float* __restrict__ out_data) {
        int num_blocks = std::min(MAX_NUM_BLOCKS_PER_KERNEL, (nelems - 1) / MAX_NUM_THREADS_PER_BLOCK + 1);
        scale<<<num_blocks, MAX_NUM_THREADS_PER_BLOCK, 0, stream>>>(nelems, data, alpha, out_data);
        return hipGetLastError();
    }
}
